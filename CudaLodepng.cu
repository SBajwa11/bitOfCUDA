#include "hip/hip_runtime.h"
//Sabraaj Bajwa 1724962

#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h" 

// rgbt - each pixel contains 4 values, to blur, all surrounding pixels need to be considered 
// don't edit t on output because that will impact transparency - hiran kept it the same for neg filter

__global__ void boxBlurFilter(unsigned char * inputImage, unsigned char * outputImage){

    int tIdy = threadIdx.y; // otherwise referred to as the threadindex within a block associated with threads - declaration of variable
    int tIdx = threadIdx.x;  // otherwise referred to as the threadindex within a block associated with threads - declaration of variable   
    
    int r;
    int g;
    int b;
    int t;
    
    int threadIndex = (threadIdx.y * blockDim.x + threadIdx.x); 
    int pixel = threadIndex * 4;  // multiply the above variable by 4 for use within inputImage and outputImage

    r = inputImage[pixel]; // obtains rgb offset value
    g = inputImage[pixel+1]; // obtains rgb offset value
    b = inputImage[pixel+2]; // obtains rgb offset value
    t = inputImage[pixel+3]; // obtains rgb offset value
    
    outputImage[pixel] = 0; // declares the value as 0 prepared for the adjusted value later on in the function after running through the for loop 
    outputImage[pixel+1] = 0; // declares the value as 0 prepared for the adjusted value later on in the function after running through the for loop 
    outputImage[pixel+2] = 0; // declares the value as 0 prepared for the adjusted value later on in the function after running through the for loop 
    outputImage[pixel+3] = t; // obtains rgb offset but we do not affect the transparency as much as instructed in the lecture 

    int count = 0; // sets variable count to 0
    int sumofr = 0; // sets variable count to 0
    int sumofg = 0; // sets variable count to 0
    int sumofb =0; // sets variable count to 0

    for(int i = -1; i <=1; ++i) {
        for(int j = -1; j <= 1; ++j) { 
            int k = tIdy + i; // adds the value of i in the for statement to threadIdx.y
            int l = tIdx + j; // adds the value of j in the for statement to threadIdx.x

            if(k < 0) continue; // if the threadIdx.y is less than 0, then continue with the for loop 
            if(k >= blockDim.y) continue; // if the threadIdx.y is greater than the size of each block, continue the loop 
            if(l < 0) continue; // if the threadIdx.x is less than 0, then continue with the for loop 
            if(l >= blockDim.x) continue; // if the threadIdx.x is greater or equal to the size of each block (x), continue

            ++count; // count will increase by 1 whilst executing line as opposed to count++ which is after the statement is executed

            int threadIndexNew = (k * blockDim.x + l); // Produces a new threadIndex using the variables above which change throughout the loop 
            int pixelNew = 4 * threadIndexNew;  // multiplies the above by 4 

            r = inputImage[pixelNew]; // assigns new value to r using newly calculated threadIndexNew / pixelNew value 
            g = inputImage[pixelNew+1]; // assigns new value to r using newly calculated threadIndexNew / pixelNew value
            b = inputImage[pixelNew+2]; // assigns new value to r using newly calculated threadIndexNew / pixelNew value

            sumofr += r; // This adds r to sumofr then assigns the overall value to sumofr
            sumofg += g; // This adds g to sumofg then assigns the overall value to sumofg
            sumofb += b; // This adds b to sumofb then assigns the overall value to sumofb

        }
    }

    outputImage[pixel] += sumofr / count; // we divide by count to obtain the average, and then assign that value to output image[pixel] plus the original value of outputimage[pixel]
    outputImage[pixel+1] += sumofg / count; // we divide by count to obtain the average, and then assign that value to output image[pixel+1] plus the original value of outputimage[pixel+1]
    outputImage[pixel+2] += sumofb / count; // we divide by count to obtain the average, and then assign that value to output image[pixel+2] plus the original value of outputimage[pixel+2]
   
}
    
    int main(int argc, char ** argv){
    
      unsigned int errorDecode; //variable will hold whether there was an issue with loading in the png file
      unsigned char* cpuImage; //this variable will hold all of our image data
      unsigned int width, height; //holds the width and height of image
      
      char * filename = argv[1]; // works as pointers for the first command line argument, so when running the program, it requires the file name and the new file name, this line is for the file name only
      char * newFilename = argv[2]; // works as pointers for the second command line argument, so when running the program, it requires the file name and the new file name, this line is for the new file name only so e.g. test.png
    
      errorDecode = lodepng_decode32_file(&cpuImage, &width, &height, filename); // (where to store the image data, width, height, which file?)
      
      if(errorDecode){
        printf("error %u: %s\n", errorDecode, lodepng_error_text(errorDecode));  // if error is found when decoding image, print error message 
      }
      
      printf("width of image is %d\nheight of image is %d\n", width, height); // States the height and width of the image, which for test purposes was the 4x4.png file so 4 by 4

      int arraySize = width*height*4; // 
      int memorySize = arraySize * sizeof(unsigned char); //sizeof(unsigned char) is multiplied by array size to produce a value stored in arraysize
      
      unsigned char cpuOutImage[arraySize];
      
      unsigned char* gpuInput; // used as a means to store character values which stores values from 0-255 since it is unsigned
      unsigned char* gpuOutput; // used as a means to store character values which stores values from 0-255 since it is unsigned
      
      hipMalloc( (void**) &gpuInput, memorySize); // cuda malloc device array, since it returns a pointer it requires a double pointer with **void
      hipMalloc( (void**) &gpuOutput, memorySize); // cuda malloc device array, since it returns a pointer it requires a double pointer with **void
      
      hipMemcpy(gpuInput, cpuImage, memorySize, hipMemcpyHostToDevice); // synchronises the kernal call with the transfer of memory 
      
      dim3 grid(1,1); // a variable of type integer vector which is used to specify dimensions, in this case 1,1 which works by the kernel executing as a grid of blocks, since the image is 4x4 it uses 4 blocks
      dim3 block(width, height); // This is a group of threads, which takes in the width and height of the image, in this case 4x4 so links with the grid function to complete boxBlurFilter below 
      boxBlurFilter<<< grid, block >>>(gpuInput, gpuOutput); // Call boxBlurFilter using the above functions with dim3, grid and block   
      hipDeviceSynchronize(); // hipDeviceSynchronize was initially used but it is since deprecated, so for program longevity device was used instead

      hipMemcpy(cpuOutImage, gpuOutput, memorySize, hipMemcpyDeviceToHost); // synchronises the kernal call with the transfer of memory 
      
      unsigned int errorEncode = lodepng_encode32_file(newFilename, cpuOutImage, width, height); // encodes the file and pushes it out with the new filename specified in the command line 
      
      if(errorEncode) {
        printf("error %u: %s\n", errorEncode, lodepng_error_text(errorEncode)); // if there is an error with encoding the file and pushing it out as a new file, it will display an error message 
      }

      hipFree(gpuInput); // returns memory for reallocation for gpuInput
      hipFree(gpuOutput); // returns memory for reallocation for gpuOutput 
    
      free(cpuImage); // deallocates memory
    
    }
    